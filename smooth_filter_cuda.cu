#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <string.h>
#include <time.h>

#include <hip/hip_runtime.h>


const size_t BMP_HEADER_SIZE = 54;

const size_t BMP_HEADER_WIDTH_OFFSET = 18;
const size_t BMP_HEADER_HEIGHT_OFFSET = 22;

const size_t PIXEL_REAL_SIZE = 3;

size_t getThreadsPerBlock(int deviceNum)
{
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, deviceNum);

	return (size_t)properties.maxThreadsPerBlock;
}

size_t getWidth(const unsigned char * header)
{
	const unsigned char * start_position = header + BMP_HEADER_WIDTH_OFFSET;
	unsigned int value = *((const unsigned int*)start_position);
	return (size_t)value;
}

size_t getHeight(const unsigned char * header)
{
	const unsigned char * start_position = header + BMP_HEADER_HEIGHT_OFFSET;
	unsigned int value = *((const unsigned int*)start_position);
	return (size_t)value;
}

struct RgbPixel
{
	unsigned char b;
	unsigned char g;
	unsigned char r;
};

struct RgbPixel * allocateMemory(size_t size)
{
	struct RgbPixel * data = (struct RgbPixel *)malloc(sizeof(struct RgbPixel) * size);
	return data;
}

struct RgbPixel * cudaAllocateMemory(size_t size)
{
	struct RgbPixel * data = NULL;
	hipError_t hipError_t = hipMalloc((void **)&data, sizeof(struct RgbPixel) * size);
	return hipError_t == hipSuccess ? data : NULL;
}

void deallocateMemory(struct RgbPixel ** data)
{
	free(*data);
	*data = NULL;
}

void cudaDeallocateMemory(struct RgbPixel ** data)
{
	hipFree(*data);
	*data = NULL;
}

bool readBmp(
	const char * filename,
	unsigned char * header,
	struct RgbPixel ** bmp
)
{
	size_t read;

	FILE * fileBmp = fopen(filename, "rb");
	if (!fileBmp)
		return false;

	read = fread(header, sizeof(unsigned char), BMP_HEADER_SIZE, fileBmp);
	if (read != BMP_HEADER_SIZE)
	{
		fclose(fileBmp);
		return false;
	}

	size_t width = getWidth(header), height = getHeight(header);
	size_t size = width * height;

	*bmp = allocateMemory(size);
	for (size_t i = 0; i < size; i++)
	{
		read = fread((*bmp) + i, sizeof(unsigned char), PIXEL_REAL_SIZE, fileBmp);
		if (read != PIXEL_REAL_SIZE)
		{
			deallocateMemory(bmp);
			fclose(fileBmp);
			return false;
		}
	}

	fclose(fileBmp);
	return true;
}

bool writeBmp(
	const char * filename,
	const unsigned char * header,
	struct RgbPixel * bmp
)
{
	FILE * bmpFile = fopen(filename, "wb");
	if (!bmpFile)
	{
		fclose(bmpFile);
		return false;
	}

	fwrite(header, sizeof(unsigned char), BMP_HEADER_SIZE, bmpFile);
	size_t width = getWidth(header), height = getHeight(header);
	size_t size = width * height;

	for (size_t i = 0; i < size; i++)
		fwrite(bmp + i, sizeof(unsigned char), PIXEL_REAL_SIZE, bmpFile);

	fclose(bmpFile);
	return true;
}

__global__
void processImageSmoothFilter(struct RgbPixel * bmp, size_t width, size_t height, size_t radius, struct RgbPixel * result)
{
#define ROUND(x) (unsigned char)((x) + 0.5)
	size_t index = blockDim.x * blockIdx.x + threadIdx.x;
	size_t i = index / width, j = index % width;

	unsigned int r = 0, g = 0, b = 0;

	size_t
		starti = (i < radius ? 0 : i - radius),
		endi = (i + radius >= height ? height - 1 : i + radius),
		startj = (j < radius ? 0 : j - radius),
		endj = (j + radius >= width ? width - 1 : j + radius);

	size_t count = (endi - starti + 1) * (endj - startj + 1);
	for (size_t ii = starti; ii <= endi; ii++)
		for (size_t jj = startj; jj <= endj; jj++)
		{
			size_t position = ii * width + jj;
			r += bmp[position].r;
			g += bmp[position].g;
			b += bmp[position].b;
		}

	result[index].r = ROUND((float)r / count);
	result[index].g = ROUND((float)g / count);
	result[index].b = ROUND((float)b / count);
#undef ROUND
}

int main(int argc, char * argv[])
{
	if (argc != 4)
	{
		printf("%s\n", "Usage: <program name> <input bmp file name> <output bmp file name> <radius>");
		return 0;
	}

	const char * inputBmpFileName = argv[1];
	const char * outputBmpFileName = argv[2];
	size_t radius = (size_t)atoi(argv[3]);

	unsigned char header[BMP_HEADER_SIZE];
	struct RgbPixel * bmp = NULL;

	time_t totalBegin = time(NULL);

	bool readResult = readBmp(inputBmpFileName, header, &bmp);
	if (!readResult)
	{
		printf("Cannot read bmp file %s\n", inputBmpFileName);
		return 1;
	}

	size_t width = getWidth(header), height = getHeight(header);
	size_t size = width * height;
	struct RgbPixel * cudaBmp = cudaAllocateMemory(size);
	struct RgbPixel * cudaResultBmp = cudaAllocateMemory(size);

	if (!cudaBmp && !cudaResultBmp)
	{
		printf("%s\n", "Cannot allocate memory in GPU");
		return 1;
	}

	hipMemcpy(cudaBmp, bmp, size * sizeof(struct RgbPixel), hipMemcpyHostToDevice);
	time_t begin = time(NULL);
	size_t threadsPerBlock = getThreadsPerBlock(0);
	size_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
	for (int i = 0; i < 100; i++)
	{
		processImageSmoothFilter<<<blocks, threadsPerBlock>>>(cudaBmp, width, height, radius, cudaResultBmp);
		hipDeviceSynchronize();
	}
	time_t end = time(NULL);

	struct RgbPixel * resultBmp = allocateMemory(size);
	hipMemcpy(resultBmp, cudaResultBmp, size * sizeof(struct RgbPixel), hipMemcpyDeviceToHost);

	bool writeResult = writeBmp(outputBmpFileName, header, resultBmp);
	if (!writeResult)
	{
		printf("Cannot write bmp file %s\n", outputBmpFileName);
		return 1;
	}

	deallocateMemory(&bmp);
	deallocateMemory(&resultBmp);

	cudaDeallocateMemory(&cudaBmp);
	cudaDeallocateMemory(&cudaResultBmp);

	time_t totalEnd = time(NULL);

	printf("Algorithm time: %.2f sec.\n", (double)(end - begin));
	printf("Total time: %.2f sec.\n", (double)(totalEnd - totalBegin));

	return 0;
}

